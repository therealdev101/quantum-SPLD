
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

#ifndef HASH_SLOT_BYTES
#define HASH_SLOT_BYTES 256
#endif
#ifndef TX_SLOT_BYTES
#define TX_SLOT_BYTES 1024
#endif
#ifndef HASH_OUTPUT_BYTES
#define HASH_OUTPUT_BYTES 32
#endif
#ifndef SIGNATURE_BYTES
#define SIGNATURE_BYTES 65
#endif
#ifndef MESSAGE_BYTES
#define MESSAGE_BYTES 32
#endif
#ifndef PUBKEY_BYTES
#define PUBKEY_BYTES 65
#endif
#ifndef TX_RESULT_BYTES
#define TX_RESULT_BYTES 64
#endif

// Exported Go helpers implemented in gpu_bridge_exports.go.
extern void go_keccak256(const uint8_t* input, int length, uint8_t* output);
extern int go_verify_signature(const uint8_t* signature, const uint8_t* message, const uint8_t* public_key);
extern int go_process_transaction(const uint8_t* tx, int length, uint8_t* output);

static inline uint32_t clamp_length(uint32_t length, uint32_t max_length) {
    return length > max_length ? max_length : length;
}

#ifdef __cplusplus
extern "C" {
#endif

int cuda_init_device() {
    // The implementation is CPU backed, but we expose at least one logical device
    // so the Go layer enables the GPU pipeline by default.
    return 1;
}

int cuda_process_hashes(void* input, void* lengths, int count, void* output) {
    if (!input || !lengths || !output || count <= 0) {
        return -1;
    }

    uint8_t* in = (uint8_t*)input;
    uint32_t* lens = (uint32_t*)lengths;
    uint8_t* out = (uint8_t*)output;

    for (int i = 0; i < count; i++) {
        uint32_t length = clamp_length(lens[i], HASH_SLOT_BYTES);
        uint8_t* item_in = in + ((size_t)i * HASH_SLOT_BYTES);
        uint8_t* item_out = out + ((size_t)i * HASH_OUTPUT_BYTES);
        go_keccak256(item_in, (int)length, item_out);
    }
    return 0;
}

int cuda_verify_signatures(void* sigs, void* msgs, void* keys, int count, void* results) {
    if (!sigs || !msgs || !keys || !results || count <= 0) {
        return -1;
    }

    uint8_t* sig_ptr = (uint8_t*)sigs;
    uint8_t* msg_ptr = (uint8_t*)msgs;
    uint8_t* key_ptr = (uint8_t*)keys;
    uint8_t* out_ptr = (uint8_t*)results;

    for (int i = 0; i < count; i++) {
        uint8_t* sig = sig_ptr + ((size_t)i * SIGNATURE_BYTES);
        uint8_t* msg = msg_ptr + ((size_t)i * MESSAGE_BYTES);
        uint8_t* key = key_ptr + ((size_t)i * PUBKEY_BYTES);
        int ok = go_verify_signature(sig, msg, key);
        out_ptr[i] = (uint8_t)(ok ? 1 : 0);
    }
    return 0;
}

int cuda_process_transactions(void* txs, void* lengths, int count, void* results) {
    if (!txs || !lengths || !results || count <= 0) {
        return -1;
    }

    uint8_t* tx_ptr = (uint8_t*)txs;
    uint32_t* lens = (uint32_t*)lengths;
    uint8_t* out_ptr = (uint8_t*)results;

    for (int i = 0; i < count; i++) {
        uint8_t* tx = tx_ptr + ((size_t)i * TX_SLOT_BYTES);
        uint32_t length = clamp_length(lens[i], TX_SLOT_BYTES);
        uint8_t* out = out_ptr + ((size_t)i * TX_RESULT_BYTES);
        // go_process_transaction fills the output buffer with metadata and returns 0 on success.
        int status = go_process_transaction(tx, (int)length, out);
        if (status != 0) {
            return status;
        }
    }
    return 0;
}

void cuda_cleanup() {
    // No resources to release in the CPU-backed implementation.
}

#ifdef __cplusplus
} // extern "C"
#endif
